#include <iostream>
#include <map>
#include <set>
#include <vector>
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

using namespace std;

#define CUDA_SAFE_CALL( err ) (safe_call(err, __LINE__))
#define MAX_THREADS_PER_BLOCK 1024

void safe_call(hipError_t ret, int line)
{
	if(ret!=hipSuccess)
	{
		printf("Error at line %d : %s\n",line,hipGetErrorString(ret));
		exit(-1);
	}
}

typedef struct __graph
{
	int E;
	int *from;
	int *to;
} graph_t;

__device__ bool d_over;

__global__ void reset()
{
	d_over = false;
}

// Print the graph
/*__global__ void temp_kernel(graph_t * graph) 
{
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	if(id == 0)
	{
		int j;
		for(j=0; j<graph->adj_prefix_sum[graph->V-1]; j++)
			printf("%d ",graph->adj[j]);
		printf("\n");
	}
}*/

__global__ void init(int * vertices, int starting_vertex, int num_vertices)
{
	int v = blockDim.x*blockIdx.x + threadIdx.x;
	if (v==starting_vertex)
		vertices[v] = 0;
	else if(v < num_vertices)
		vertices[v] = -1;
}

__global__ void bfs(const graph_t * graph, int * vertices, int current_depth)
{
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	if(id < graph->E)
	{
		int f = graph->from[id];
		if(vertices[f] == current_depth)
		{
			int e = graph->to[id];
			if(vertices[e] == -1)
			{
				vertices[e] = current_depth+1;
				d_over = true;
			}
		}
	}
}

int main(int argc, char * argv[])
{
	static char * filename;
	if(argc>2)
	{
		printf("./a.out <filename>\n");
		exit(-1);
	}
	else if(argc==2)
	{
		filename = argv[1];
	}
	else
	{
		filename = "../data/input.txt";
	}

	FILE * fp = fopen(filename,"r");
	if(!fp)
	{
		printf("Error reading file.\n");
		exit(-1);
	}

	/* Set cuda device to K40  */
	CUDA_SAFE_CALL(hipSetDevice(0));

	/* Get graph from file into CPU memory  */
	int num_vertices, num_edges, i, j;
	fscanf(fp,"%d %d",&num_vertices,&num_edges);

	graph_t *graph_host;
	CUDA_SAFE_CALL(hipMallocManaged((void **)&graph_host, sizeof(graph_t)));

	graph_host->E = num_edges;

	CUDA_SAFE_CALL(hipMallocManaged((void **)&(graph_host->from), num_edges*sizeof(int)));

	CUDA_SAFE_CALL(hipMallocManaged((void **)&(graph_host->to), num_edges*sizeof(int *)));

	set<int> vertices;
	vector< pair<int,int> > edges;
	int s,d;
	for(i=0; i<num_edges; i++)
	{
		fscanf(fp,"%d",&s);
		fscanf(fp,"%d",&d);
		vertices.insert(s);
		vertices.insert(d);
		edges.push_back(make_pair(s,d));
	}

	sort(edges.begin(),edges.end());

	i=0;
	//int l=0,r=0;
	//set<int>::iterator fe=vertices.begin();
	//set<int>::iterator se=vertices.begin();
	for(vector< pair<int,int> >::iterator it = edges.begin() ; it != edges.end(); ++it)
	{
	/*	while((*fe)!=(*it).first && fe!=vertices.end()) 
		{
			l++;
			se = vertices.begin();
			r=0;
		}
		while((*se)!=(*it).second && se!=vertices.end())
		{
			r++;
		}
		*/
		int l = distance(vertices.begin(),vertices.find((*it).first)); // C++ set stores in sorted order by default
		int r = distance(vertices.begin(),vertices.find((*it).second));

		graph_host->from[i]=l;
		graph_host->to[i]=r;
		i++;
	}


	/*****************************************************
	XXX: GPU does not know the size of each adjacency list.
	For that, a new struct containing size of list and list 
	has to be created and passed to GPU memory. Too much hassle.

	OR

	Create 1-D array in the graph itself which contains the 
	size of each list.
	*****************************************************/

	//temp_kernel<<<1,1>>>(graph_device);

	int num_of_blocks = 1;
	int num_of_threads_per_block = num_edges;

	if(num_edges > MAX_THREADS_PER_BLOCK)
	{
		num_of_blocks = (int)ceil(num_edges/(double)MAX_THREADS_PER_BLOCK); 
		num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}

	int * vertices_host;
	CUDA_SAFE_CALL(hipMallocManaged((void **)&vertices_host, num_vertices*sizeof(int)));

	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);
	
	hipEvent_t start,end;
	float diff;
	double time = 0;

	CUDA_SAFE_CALL(hipEventCreate(&start));
	CUDA_SAFE_CALL(hipEventCreate(&end));

	init<<<grid,threads>>> (vertices_host, 0, num_vertices);

	bool stop;
	int k=0;
	do
	{
		stop = false;
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_over), &stop, sizeof(bool),0, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipDeviceSynchronize());

		CUDA_SAFE_CALL(hipEventRecord(start,0));
					
		bfs<<<grid, threads>>> (graph_host, vertices_host, k);
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		CUDA_SAFE_CALL(hipEventRecord(end,0));
		CUDA_SAFE_CALL(hipEventSynchronize(end));
		CUDA_SAFE_CALL(hipEventElapsedTime(&diff, start, end));
		time += diff*1.0e-3;

		CUDA_SAFE_CALL(hipMemcpyFromSymbol(&stop, HIP_SYMBOL(d_over), sizeof(bool),0, hipMemcpyDeviceToHost));
		k++;
	}while(stop);

	printf("Number of iterations : %d\n",k);
	for(int i = 0; i < num_vertices; i++)
	{
		printf("Vertex %d Distance %d\n",i,vertices_host[i]);
	}
	printf("Time: %f ms\n",time);

	CUDA_SAFE_CALL(hipFree(vertices_host));
	CUDA_SAFE_CALL(hipFree(graph_host->from));
	CUDA_SAFE_CALL(hipFree(graph_host->to));
	CUDA_SAFE_CALL(hipFree(graph_host));

	CUDA_SAFE_CALL(hipEventDestroy(start));
	CUDA_SAFE_CALL(hipEventDestroy(end));

	return 0;
}
