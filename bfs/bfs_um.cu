#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CUDA_SAFE_CALL( err ) (safe_call(err, __LINE__))
#define MAX_THREADS_PER_BLOCK 1024

void safe_call(hipError_t ret, int line)
{
	if(ret!=hipSuccess)
	{
		printf("Error at line %d : %s\n",line,hipGetErrorString(ret));
		exit(-1);
	}
}

typedef struct __graph
{
	int V;
	int *adj_prefix_sum;
	int *adj;
} graph_t;

__device__ bool d_over;

__global__ void reset()
{
	d_over = false;
}

__global__ void temp_kernel(graph_t * graph) 
{
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	if(id == 0)
	{
		int j;
		for(j=0; j<graph->adj_prefix_sum[graph->V-1]; j++)
			printf("%d ",graph->adj[j]);
		printf("\n");
	}
}

__global__ void init(int * vertices, int starting_vertex, int num_vertices)
{
	int v = blockDim.x*blockIdx.x + threadIdx.x;
	if (v==starting_vertex)
		vertices[v] = 0;
	else
		vertices[v] = -1;
}

__global__ void bfs(const graph_t * graph, int * vertices, int current_depth)
{
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	if(id < graph->V)
	{
		if(vertices[id] == current_depth)
		{
			int i;
			if(id == 0) 
				i = 0;
			else
				i = graph->adj_prefix_sum[id-1];
			for(; i < graph->adj_prefix_sum[id]; i++)
			{
				if(vertices[graph->adj[i]] == -1)
				{
					vertices[graph->adj[i]] = current_depth+1;
					d_over = true;
				}
			}
		}
	}
}

int main(int argc, char * argv[])
{
	static char * filename;
	if(argc>2)
	{
		printf("./a.out <filename>\n");
		exit(-1);
	}
	else if(argc==2)
	{
		filename = argv[1];
	}
	else
	{
		filename = "input.txt";
	}

	FILE * fp = fopen(filename,"r");
	if(!fp)
	{
		printf("Error reading file.\n");
		exit(-1);
	}

	/* Set cuda device to K40  */
	CUDA_SAFE_CALL(hipSetDevice(0));

	/* Get graph from file into CPU memory  */
	int num_vertices, num_edges, i, j;
	fscanf(fp,"%d %d",&num_vertices,&num_edges);

	graph_t *graph_host;
	CUDA_SAFE_CALL(hipMallocManaged((void **)&graph_host, sizeof(graph_t)));

	graph_host->V = num_vertices;

	CUDA_SAFE_CALL(hipMallocManaged((void **)&(graph_host->adj_prefix_sum), num_vertices*sizeof(int)));

	CUDA_SAFE_CALL(hipMallocManaged((void **)&(graph_host->adj), num_edges*sizeof(int *)));

	for(i=0; i<num_vertices; i++)
	{
		int edges_per_vertex;
		fscanf(fp,"%d",&edges_per_vertex);
		if(i>0)
		{
			graph_host->adj_prefix_sum[i] = graph_host->adj_prefix_sum[i-1]+edges_per_vertex;
			j = graph_host->adj_prefix_sum[i-1];
		}
		else
		{
			graph_host->adj_prefix_sum[i] = edges_per_vertex;
			j = 0;
		}
		for(; j<graph_host->adj_prefix_sum[i]; j++)
		{
			fscanf(fp,"%d",&graph_host->adj[j]);
		}
	}

	/*****************************************************
	XXX: GPU does not know the size of each adjacency list.
	For that, a new struct containing size of list and list 
	has to be created and passed to GPU memory. Too much hassle.

	OR

	Create 1-D array in the graph itself which contains the 
	size of each list.
	*****************************************************/

	//temp_kernel<<<1,1>>>(graph_device);

	int num_of_blocks = 1;
	int num_of_threads_per_block = num_vertices;

	if(num_vertices>MAX_THREADS_PER_BLOCK)
	{
		num_of_blocks = (int)ceil(num_vertices/(double)MAX_THREADS_PER_BLOCK); 
		num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}

	int * vertices_host;
	CUDA_SAFE_CALL(hipMallocManaged((void **)&vertices_host, num_vertices*sizeof(int)));

	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);
	
	hipEvent_t start,end;
	float diff;
	double time = 0;

	CUDA_SAFE_CALL(hipEventCreate(&start));
	CUDA_SAFE_CALL(hipEventCreate(&end));

	init<<<grid,threads>>> (vertices_host, 0, num_vertices);

	bool stop;
	int k=0;
	do
	{
		stop = false;
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_over), &stop, sizeof(bool),0, hipMemcpyHostToDevice));
		reset<<<1,1>>>();
		CUDA_SAFE_CALL(hipDeviceSynchronize());

		CUDA_SAFE_CALL(hipEventRecord(start,0));
					
		bfs<<<grid, threads>>> (graph_host, vertices_host, k);
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		CUDA_SAFE_CALL(hipEventRecord(end,0));
		CUDA_SAFE_CALL(hipEventSynchronize(end));
		CUDA_SAFE_CALL(hipEventElapsedTime(&diff, start, end));
		time += diff*1.0e-3;

		CUDA_SAFE_CALL(hipMemcpyFromSymbol(&stop, HIP_SYMBOL(d_over), sizeof(bool),0, hipMemcpyDeviceToHost));
		k++;
	}while(stop);

	printf("Number of iterations : %d\n",k);
	for(int i = 0; i < num_vertices; i++)
	{
		printf("Vertex %d Distance %d\n",i,vertices_host[i]);
	}
	printf("Time: %f ms\n",time);

	CUDA_SAFE_CALL(hipFree(vertices_host));
	CUDA_SAFE_CALL(hipFree(graph_host->adj));
	CUDA_SAFE_CALL(hipFree(graph_host->adj_prefix_sum));
	CUDA_SAFE_CALL(hipFree(graph_host));

	CUDA_SAFE_CALL(hipEventDestroy(start));
	CUDA_SAFE_CALL(hipEventDestroy(end));

	return 0;
}
