#include <iostream>
#include <vector>
#include <set>
#include <map>
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

using namespace std;

#define CUDA_SAFE_CALL( err ) (safe_call(err, __LINE__))
#define MAX_THREADS_PER_BLOCK 1024
#define GLOBAL_MAX_EDGES_PER_SHARD 33554432 

void safe_call(hipError_t ret, int line)
{
    if(ret!=hipSuccess)
    {
        printf("Error at line %d : %s\n",line,hipGetErrorString(ret));
        exit(-1);
    }
}

typedef struct __interval
{
    int start;
    int end;
} interval_t;

typedef struct __edge
{
    int src;
    int dest;
    int val;
} edge_t;

typedef struct __vertex
{
    int val;
} vertex_t;

typedef struct __shard
{
    int E;
    int Vstart;
    int Vend;
    int * vmap;
    vertex_t * from;
    vertex_t * to;
} shard_t;

__device__ bool d_over;

__global__ void reset()
{
    d_over = false;
}

__global__ void init(vertex_t * vertices, int starting_vertex, int num_vertices)
{
    int v = blockDim.x*blockIdx.x + threadIdx.x;
    if (v==starting_vertex)
        vertices[v].val = 0;
    else if(v < num_vertices)
        vertices[v].val = -1;
}

/*__global__ void gather_bfs(shard_t * shard, vertex_t * vertices, int current_depth)
  {
  int id = blockDim.x*blockIdx.x + threadIdx.x;
  if(id < shard->E)
  {
  if(shard->edges[id].val == (current_depth+1))
  {
  int t=shard->edges[id].dest;
  if(vertices[t].val == -1)
  {
  vertices[t].val = current_depth+1;
  d_over = true;
  }
  }
  }
  }*/

__global__ void scatter_bfs_edge(const shard_t * shard, vertex_t * vertices, int current_depth)
{
    int id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id < shard->E)
    {
        int s=shard->from[id].val;
        int t=vertices[s].val;
        if(t==current_depth)
        {
            int u=shard->to[id].val;
            if(vertices[u].val == -1)
            {
                vertices[u].val = t+1;
                d_over = true;
            }
        }
    }
}

__global__ void scatter_bfs_vertex(const shard_t * shard, vertex_t * vertices, int current_depth)
{
    int id = blockDim.x*blockIdx.x + threadIdx.x;
    int vid = id + shard->Vstart;
    if(vid <= shard->Vend)
    {
        if(vertices[vid].val == current_depth)
        {
            int i;
            if(id == 0) 
                i = 0;
            else
                i = shard->vmap[id-1];
            for(; i < shard->vmap[id]; i++)
            {
                if(vertices[shard->to[i].val].val == -1)
                {
                    vertices[shard->to[i].val].val = current_depth+1;
                    d_over = true;
                }
            }
        }
    }
}

bool cost(const edge_t &a, const edge_t &b)
{
    return ((a.src < b.src) || (a.src == b.src && a.dest < b.dest));
}

int main(int argc, char * argv[])
{
    struct timeval t1,t2;
    static char * filename;
    if(argc!=2)
    {
        printf("./a.out <filename>\n");
        exit(-1);
    }
    else
    {
        filename = argv[1];
    }

    FILE * fp = fopen(filename,"r");
    if(!fp)
    {
        printf("Error reading file.\n");
        exit(-1);
    }

    /* Set cuda device to K40  */
    CUDA_SAFE_CALL(hipSetDevice(0));

    printf("Begin file reading...\n");

    /* Get graph from file into CPU memory  */
    int num_vertices, num_edges, i, j, k;
    fscanf(fp,"%d %d",&num_vertices,&num_edges);

    //We are always going to have atleast 2 shards to have double bufferring
    int ns = num_edges / GLOBAL_MAX_EDGES_PER_SHARD;
    int MAX_EDGES_PER_SHARD = (ns == 0) ? (num_edges + 1)/2 : (num_edges + 1)/(ns + 1); //We do this to balance the no of edges in the shards

    //Array of vectors. vector i contains the in edges of vertex i
    vector< vector<edge_t> > outEdges(num_vertices);
    int * prefixV = (int *) calloc(num_vertices,sizeof(int));
    int s,d,v;

    // In Graphchi case, I am storing the source depth in each edge
    // In X-stream case, I am storing the destination depth in each edge
    for(i=0; i<num_edges; i++)
    {
        fscanf(fp,"%d",&s);
        fscanf(fp,"%d",&d);
        edge_t e;
        e.src=s;
        e.dest=d;
        outEdges[s].push_back(e);
    }
    printf("Finished file reading.\n");

    printf("\nBegin interval construction...\n");

    // Construction of intervals
    gettimeofday(&t1,NULL);
    int num_intervals = 0, add = 1;
    vector<int> startInter;
    prefixV[0] = outEdges[0].size();
    if(prefixV[0] > MAX_EDGES_PER_SHARD)
    {
        startInter.push_back(0);
        num_intervals++;
        add = 0;
    }
    for(i=1; i<num_vertices; i++)
    {
        prefixV[i] = outEdges[i].size();    
        if(add==1)
            prefixV[i] += prefixV[i-1];
        if(prefixV[i] > MAX_EDGES_PER_SHARD)
        {
            startInter.push_back(i);
            num_intervals++;
            add = 0;
        }
        else
            add = 1;
    }
    if(add==1)
    {
        startInter.push_back(i-1);
        num_intervals++;
    }


    interval_t * interval = (interval_t *) malloc(num_intervals*sizeof(interval_t));
    for(i=0; i<num_intervals; i++)
    {
        interval[i].start = (i == 0) ? 0 : (startInter[i-1]+1);
        interval[i].end = startInter[i];
    }
    gettimeofday(&t2,NULL);
    printf("Time to construct intervals : %f sec\n",((t2.tv_sec+t2.tv_usec*1.0e-6)-(t1.tv_sec+t1.tv_usec*1.0e-6)));


    printf("\nBegin shard construction...\n");
    //Construction of shard
    gettimeofday(&t1,NULL);
    shard_t * shard = (shard_t *) malloc(num_intervals*sizeof(shard_t));

    //Finding the max number of edges in a shard
    // We will allocate space for that many edges to each shard to maintain consistency
    int MAX_NUM_EDGES_SHARD = INT_MIN;
    int MAX_NUM_VERTICES_SHARD = INT_MIN;

    for(i=0; i<num_intervals; i++)
    {
        int t = prefixV[interval[i].end];
        if(t > MAX_NUM_EDGES_SHARD)
            MAX_NUM_EDGES_SHARD = t;
        int q = interval[i].end-interval[i].start+1;
        if(q > MAX_NUM_VERTICES_SHARD)
            MAX_NUM_VERTICES_SHARD = q;
    }

    for(i=0; i<num_intervals; i++)
    {
        // first and last vertices in shard
        shard[i].Vstart = interval[i].start;
        shard[i].Vend = interval[i].end;
        shard[i].E = prefixV[interval[i].end]; 

        shard[i].vmap = (int *) malloc(MAX_NUM_VERTICES_SHARD*sizeof(int));
        shard[i].from = (vertex_t *) malloc(MAX_NUM_EDGES_SHARD*sizeof(vertex_t));
        shard[i].to = (vertex_t *) malloc(MAX_NUM_EDGES_SHARD*sizeof(vertex_t));
    }


    for(i=0; i<num_intervals; i++)
    {
        vector<edge_t> tempEdges;
        for(j=interval[i].start; j<=interval[i].end; j++)
        {
            for(vector<edge_t>::iterator it=outEdges[j].begin(); it!=outEdges[j].end(); ++it)
                tempEdges.push_back(*it);
        }

        //Sorting based on src vertex to align the edges such that the access of vertices[src] is sequential
        sort(tempEdges.begin(),tempEdges.end(),cost);

        vector< vector<edge_t> > bucket(MAX_NUM_VERTICES_SHARD);
        for (vector<edge_t>::iterator it = tempEdges.begin() ; it != tempEdges.end(); ++it)
        {
            bucket[(*it).src-interval[i].start].push_back(*it);
        }
        for(j=0;j<MAX_NUM_VERTICES_SHARD;j++)
        {
            shard[i].vmap[j] = bucket[j].size();
        }
        for(j=1;j<MAX_NUM_VERTICES_SHARD;j++)
        {
            shard[i].vmap[j] += shard[i].vmap[j-1];
        }
        k=0;
        for(j=0;j<MAX_NUM_VERTICES_SHARD;j++)
        {
            for (vector<edge_t>::iterator it = bucket[j].begin() ; it != bucket[j].end(); ++it)
            {
                shard[i].from[k].val = (*it).src;
                shard[i].to[k].val = (*it).dest;
                k++;
            }
        }
    }
    gettimeofday(&t2,NULL);
    printf("Time to construct shards : %f sec\n",((t2.tv_sec+t2.tv_usec*1.0e-6)-(t1.tv_sec+t1.tv_usec*1.0e-6)));

    hipStream_t * str;
    hipEvent_t * start;
    hipEvent_t * stop;
    int num_evts=2;
    str = (hipStream_t *) malloc(num_evts * sizeof(hipStream_t));
    start = (hipEvent_t *) malloc(num_evts * sizeof(hipEvent_t));
    stop = (hipEvent_t *) malloc(num_evts * sizeof(hipEvent_t));
    for(int i = 0; i < num_evts; i++)
    {
        CUDA_SAFE_CALL(hipStreamCreate(&(str[i])));
        CUDA_SAFE_CALL(hipEventCreate(&(start[i])));
        CUDA_SAFE_CALL(hipEventCreate(&(stop[i])));
    }

    // It will contain the visited status of each vertex
    vertex_t *vertices;
    //CUDA_SAFE_CALL(cudaMallocHost((void **)&vertices, num_vertices*sizeof(vertex_t)));
    vertex_t *vertices_host = (vertex_t *) malloc(num_vertices*sizeof(vertex_t));
    CUDA_SAFE_CALL(hipMalloc((void **)&vertices, num_vertices*sizeof(vertex_t)));

    init<<<((num_vertices+MAX_THREADS_PER_BLOCK-1)/MAX_THREADS_PER_BLOCK),MAX_THREADS_PER_BLOCK>>> (vertices, 0, num_vertices);

    float * diff = (float *) malloc(num_intervals*sizeof(float));
    double time = 0;

    // For vertex centric algo
    shard_t * shard_dev;
    int * vmap_dev;
    vertex_t * from_dev;
    vertex_t * to_dev;
    CUDA_SAFE_CALL(hipMalloc((void **)&shard_dev, sizeof(shard_t)));
    CUDA_SAFE_CALL(hipMalloc((void **)&vmap_dev, MAX_NUM_VERTICES_SHARD*sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void **)&from_dev, MAX_NUM_EDGES_SHARD*sizeof(vertex_t)));
    CUDA_SAFE_CALL(hipMalloc((void **)&to_dev, MAX_NUM_EDGES_SHARD*sizeof(vertex_t)));

    //Extra Buffer for doing double bufferring
    shard_t * shard_dev2;
    int * vmap_dev2;
    vertex_t * from_dev2;
    vertex_t * to_dev2;
    CUDA_SAFE_CALL(hipMalloc((void **)&shard_dev2, sizeof(shard_t)));
    CUDA_SAFE_CALL(hipMalloc((void **)&vmap_dev2, MAX_NUM_VERTICES_SHARD*sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void **)&from_dev2, MAX_NUM_EDGES_SHARD*sizeof(vertex_t)));
    CUDA_SAFE_CALL(hipMalloc((void **)&to_dev2, MAX_NUM_EDGES_SHARD*sizeof(vertex_t)));

    int num_of_blocks = 1;
    //int MAX_THREADS = MAX_NUM_VERTICES_SHARD;
    int MAX_THREADS = MAX_NUM_EDGES_SHARD;
    int num_of_threads_per_block = MAX_THREADS;

    if(MAX_THREADS>MAX_THREADS_PER_BLOCK)
    {
        num_of_blocks = (int)ceil(MAX_THREADS/(double)MAX_THREADS_PER_BLOCK); 
        num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
    }

    dim3  grid( num_of_blocks, 1, 1);
    dim3  threads( num_of_threads_per_block, 1, 1);

    printf("Begin kernel\n");

    int pingpong;
    bool over;
    k=0;
    do
    {
        over = false;
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_over), &over, sizeof(bool),0, hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipDeviceSynchronize());

        pingpong=0;

        for(i=0; i<num_intervals; i++)
        {
            if(pingpong==0)
            {
                //Copy Ping
                CUDA_SAFE_CALL(hipMemcpyAsync(shard_dev, &shard[i], sizeof(shard_t),hipMemcpyHostToDevice,str[0]));
                CUDA_SAFE_CALL(hipMemcpyAsync(vmap_dev, shard[i].vmap, MAX_NUM_VERTICES_SHARD*sizeof(int),hipMemcpyHostToDevice,str[0]));
                CUDA_SAFE_CALL(hipMemcpyAsync(from_dev, shard[i].from, MAX_NUM_EDGES_SHARD*sizeof(vertex_t),hipMemcpyHostToDevice,str[0]));
                CUDA_SAFE_CALL(hipMemcpyAsync(to_dev, shard[i].to, MAX_NUM_EDGES_SHARD*sizeof(vertex_t),hipMemcpyHostToDevice,str[0]));
                CUDA_SAFE_CALL(hipMemcpyAsync(&(shard_dev->vmap), &vmap_dev, sizeof(int *),hipMemcpyHostToDevice,str[0]));
                CUDA_SAFE_CALL(hipMemcpyAsync(&(shard_dev->from), &from_dev, sizeof(vertex_t *),hipMemcpyHostToDevice,str[0]));
                CUDA_SAFE_CALL(hipMemcpyAsync(&(shard_dev->to), &to_dev, sizeof(vertex_t *),hipMemcpyHostToDevice,str[0]));


                if(i>0)
                {
                    //Process Pong
                    CUDA_SAFE_CALL(hipEventRecord(start[1],str[1]));
                    scatter_bfs_edge<<<grid, threads,0,str[1]>>> (shard_dev2, vertices, k);
                    CUDA_SAFE_CALL(hipStreamSynchronize(str[1]));
                    CUDA_SAFE_CALL(hipEventRecord(stop[1],str[1]));
                    CUDA_SAFE_CALL(hipEventSynchronize(stop[1]));
                    CUDA_SAFE_CALL(hipEventElapsedTime(&diff[i-1],start[1],stop[1]));

                }

                pingpong=1;
            }
            else
            {
                //Copy Pong
                CUDA_SAFE_CALL(hipMemcpyAsync(shard_dev2, &shard[i], sizeof(shard_t),hipMemcpyHostToDevice,str[1]));
                CUDA_SAFE_CALL(hipMemcpyAsync(vmap_dev2, shard[i].vmap, MAX_NUM_VERTICES_SHARD*sizeof(int),hipMemcpyHostToDevice,str[1]));
                CUDA_SAFE_CALL(hipMemcpyAsync(from_dev2, shard[i].from, MAX_NUM_EDGES_SHARD*sizeof(vertex_t),hipMemcpyHostToDevice,str[1]));
                CUDA_SAFE_CALL(hipMemcpyAsync(to_dev2, shard[i].to, MAX_NUM_EDGES_SHARD*sizeof(vertex_t),hipMemcpyHostToDevice,str[1]));
                CUDA_SAFE_CALL(hipMemcpyAsync(&(shard_dev2->vmap), &vmap_dev2, sizeof(int *),hipMemcpyHostToDevice,str[1]));
                CUDA_SAFE_CALL(hipMemcpyAsync(&(shard_dev2->from), &from_dev2, sizeof(vertex_t *),hipMemcpyHostToDevice,str[1]));
                CUDA_SAFE_CALL(hipMemcpyAsync(&(shard_dev2->to), &to_dev2, sizeof(vertex_t *),hipMemcpyHostToDevice,str[1]));

                //Process Pong
                CUDA_SAFE_CALL(hipEventRecord(start[0],str[0]));
                scatter_bfs_edge<<<grid, threads,0,str[0]>>> (shard_dev, vertices, k);
                CUDA_SAFE_CALL(hipStreamSynchronize(str[0]));
                CUDA_SAFE_CALL(hipEventRecord(stop[0],str[0]));
                CUDA_SAFE_CALL(hipEventSynchronize(stop[0]));
                CUDA_SAFE_CALL(hipEventElapsedTime(&diff[i-1],start[0],stop[0]));

                pingpong=0;
            }
        }
        if(pingpong==0)
        {
            //Process Pong
            CUDA_SAFE_CALL(hipEventRecord(start[1],str[1]));
            scatter_bfs_edge<<<grid, threads,0,str[1]>>> (shard_dev2, vertices, k);
            CUDA_SAFE_CALL(hipStreamSynchronize(str[1]));
            CUDA_SAFE_CALL(hipEventRecord(stop[1],str[1]));
            CUDA_SAFE_CALL(hipEventSynchronize(stop[1]));
            CUDA_SAFE_CALL(hipEventElapsedTime(&diff[i-1],start[1],stop[1]));
        }
        else
        {
            //Process Pong
            CUDA_SAFE_CALL(hipEventRecord(start[0],str[0]));
            scatter_bfs_edge<<<grid, threads,0,str[0]>>> (shard_dev, vertices, k);
            CUDA_SAFE_CALL(hipStreamSynchronize(str[0]));
            CUDA_SAFE_CALL(hipEventRecord(stop[0],str[0]));
            CUDA_SAFE_CALL(hipEventSynchronize(stop[0]));
            CUDA_SAFE_CALL(hipEventElapsedTime(&diff[i-1],start[1],stop[1]));
        }

        for(i=0;i<num_intervals;i++)
            time += diff[i];

        CUDA_SAFE_CALL(hipMemcpyFromSymbol(&over, HIP_SYMBOL(d_over), sizeof(bool),0, hipMemcpyDeviceToHost));
        k++;
    }while(over);

    printf("Number of iterations : %d\n",k);
/*      CUDA_SAFE_CALL(cudaMemcpy(vertices_host, vertices, num_vertices*sizeof(vertex_t), cudaMemcpyDeviceToHost));
        for(int i = 0; i < num_vertices; i++)
        {
        printf("Vertex %d Distance %d\n",i,vertices_host[i].val);
        }
    */
    printf("Time: %f ms\n",time);

    for(int i = 0; i < num_evts; i++)
    {
        CUDA_SAFE_CALL(hipStreamDestroy(str[i]));
        CUDA_SAFE_CALL(hipEventDestroy(start[i]));
        CUDA_SAFE_CALL(hipEventDestroy(stop[i]));
    }

    free(interval);
    for(i=0; i<num_intervals; i++)
    {
        free(shard[i].vmap);
        free(shard[i].from);
        free(shard[i].to);
    }
    free(shard);
    free(vertices_host);
    CUDA_SAFE_CALL(hipFree(vertices));
    CUDA_SAFE_CALL(hipFree(vmap_dev));
    CUDA_SAFE_CALL(hipFree(from_dev));
    CUDA_SAFE_CALL(hipFree(to_dev));
    CUDA_SAFE_CALL(hipFree(shard_dev));
    CUDA_SAFE_CALL(hipFree(vmap_dev2));
    CUDA_SAFE_CALL(hipFree(from_dev2));
    CUDA_SAFE_CALL(hipFree(to_dev2));
    CUDA_SAFE_CALL(hipFree(shard_dev2));

    return 0;
}
