#include <iostream>
#include <vector>
#include <set>
#include <map>
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

using namespace std;

#define CUDA_SAFE_CALL( err ) (safe_call(err, __LINE__))
#define MAX_THREADS_PER_BLOCK 1024
#define MAX_EDGES_PER_SHARD 33554432 

void safe_call(hipError_t ret, int line)
{
	if(ret!=hipSuccess)
	{
		printf("Error at line %d : %s\n",line,hipGetErrorString(ret));
		exit(-1);
	}
}

typedef struct __interval
{
	int start;
	int end;
} interval_t;

typedef struct __edge
{
	int src;
	int dest;
	int val;
} edge_t;

typedef struct __vertex
{
	int val;
} vertex_t;

typedef struct __shard
{
	int E;
	int Vstart;
	int Vend;
	edge_t * edges;
} shard_t;
/*
typedef struct __graph
{
	vertex_t * vertices;
} graph_t;

graph_t * load_subgraph(interval_t, vector<edge_t>);
*/
__device__ bool d_over;

__global__ void reset()
{
	d_over = false;
}

__global__ void init(vertex_t * vertices, int starting_vertex, int num_vertices)
{
	int v = blockDim.x*blockIdx.x + threadIdx.x;
	if (v==starting_vertex)
		vertices[v].val = 0;
	else if(v < num_vertices)
		vertices[v].val = -1;
}

/*__global__ void gather_bfs(shard_t * shard, vertex_t * vertices, int current_depth)
{
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	if(id < shard->E)
	{
		if(shard->edges[id].val == (current_depth+1))
		{
			int t=shard->edges[id].dest;
			if(vertices[t].val == -1)
			{
				vertices[t].val = current_depth+1;
				d_over = true;
			}
		}
	}
}*/

__global__ void scatter_bfs(const shard_t * shard, vertex_t * vertices, int current_depth, int V)
{
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	if(id < shard->E)
	{
		int s=shard->edges[id].src;
		if(s < V)
		{
			int t=vertices[s].val;
			if(t==current_depth)
			{
				//shard->edges[id].val = t+1;
				int u=shard->edges[id].dest;
				if(u < V)
				{
					if(vertices[u].val == -1)
					{
						vertices[u].val = t+1;
						d_over = true;
					}
				}
				else
					printf("Illegal vertex dest: %d\n",u);
			}
		}
		else
			printf("Illegal vertex src: %d\n",s);
	}
}


bool cost(const edge_t &a, const edge_t &b)
{
	    return (a.src < b.src);
}

int main(int argc, char * argv[])
{
	struct timeval t1,t2;
	static char * filename;
	if(argc!=2)
	{
		printf("./a.out <filename>\n");
		exit(-1);
	}
	else
	{
		filename = argv[1];
	}

	FILE * fp = fopen(filename,"r");
	if(!fp)
	{
		printf("Error reading file.\n");
		exit(-1);
	}

	/* Set cuda device to K40  */
	CUDA_SAFE_CALL(hipSetDevice(0));

	printf("Begin file reading...\n");

	/* Get graph from file into CPU memory  */
	int num_vertices, num_edges, i, j, k;
	fscanf(fp,"%d %d",&num_vertices,&num_edges);

	
	//Array of vectors. vector i contains the in edges of vertex i
	vector< vector<edge_t> > outEdges(num_vertices);
	int * prefixV = (int *) calloc(num_vertices,sizeof(int));
	int s,d,v;

	// In Graphchi case, I am storing the source depth in each edge
	// In X-stream case, I am storing the destination depth in each edge
	for(i=0; i<num_edges; i++)
	{
		fscanf(fp,"%d",&s);
		fscanf(fp,"%d",&d);
		edge_t e;
		e.src=s;
		e.dest=d;
		outEdges[s].push_back(e);
	}
	printf("Finished file reading.\n");
	
	printf("\nBegin interval construction...\n");

	// Construction of intervals
	gettimeofday(&t1,NULL);
	int num_intervals = 0, add = 1;
	vector<int> startInter;
	prefixV[0] = outEdges[0].size();
	if(prefixV[0] > MAX_EDGES_PER_SHARD)
	{
		startInter.push_back(0);
		num_intervals++;
		add = 0;
	}
	for(i=1; i<num_vertices; i++)
	{
		prefixV[i] = outEdges[i].size();	
		if(add==1)
			prefixV[i] += prefixV[i-1];
		if(prefixV[i] > MAX_EDGES_PER_SHARD)
		{
			startInter.push_back(i);
			num_intervals++;
			add = 0;
		}
		else
			add = 1;
	}
	if(add==1)
	{
		startInter.push_back(i-1);
		num_intervals++;
	}


	interval_t * interval = (interval_t *) malloc(num_intervals*sizeof(interval_t));
	for(i=0; i<num_intervals; i++)
	{
		interval[i].start = (i == 0) ? 0 : (startInter[i-1]+1);
		interval[i].end = startInter[i];
	}
	gettimeofday(&t2,NULL);
	printf("Time to construct intervals : %f sec\n",((t2.tv_sec+t2.tv_usec*1.0e-6)-(t1.tv_sec+t1.tv_usec*1.0e-6)));


	printf("\nBegin shard construction...\n");
	//Construction of shards
	gettimeofday(&t1,NULL);
	shard_t * shard_host = (shard_t *) malloc(num_intervals*sizeof(shard_t));

	//Finding the max number of edges in a shard
	// We will allocate space for that many edges to each shard to maintain consistency
	int MAX_NUM_EDGES_SHARD = INT_MIN;
	for(i=0; i<num_intervals; i++)
	{
		int t = prefixV[interval[i].end];
		if(t > MAX_NUM_EDGES_SHARD)
			MAX_NUM_EDGES_SHARD = t;
	}

	for(i=0; i<num_intervals; i++)
	{
		// first and last vertices in shard
		shard_host[i].Vstart = interval[i].start;
		shard_host[i].Vend = interval[i].end;

		// number of edges in shard
		shard_host[i].E = prefixV[interval[i].end];
		shard_host[i].edges = (edge_t *) malloc(MAX_NUM_EDGES_SHARD*sizeof(edge_t));
	}


	for(i=0; i<num_intervals; i++)
	{
		vector<edge_t> tempEdges;
		for(j=interval[i].start; j<=interval[i].end; j++)
		{
			for(vector<edge_t>::iterator it=outEdges[j].begin(); it!=outEdges[j].end(); ++it)
				tempEdges.push_back(*it);
		}

		//Sorting based on src vertex to align the edges such that the access of vertices[src] is sequential
		sort(tempEdges.begin(),tempEdges.end(),cost);
		j=0;
		for (vector<edge_t>::iterator it = tempEdges.begin() ; it != tempEdges.end(); ++it)
		{
			shard_host[i].edges[j] = (*it);
			j++;
		}
	}
	gettimeofday(&t2,NULL);
	printf("Time to construct shards : %f sec\n",((t2.tv_sec+t2.tv_usec*1.0e-6)-(t1.tv_sec+t1.tv_usec*1.0e-6)));

	int num_of_blocks = 1;
	int num_of_threads_per_block = MAX_NUM_EDGES_SHARD;

	if(MAX_NUM_EDGES_SHARD>MAX_THREADS_PER_BLOCK)
	{
		num_of_blocks = (int)ceil(MAX_NUM_EDGES_SHARD/(double)MAX_THREADS_PER_BLOCK); 
		num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}

	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);

	shard_t *shard;
	//CUDA_SAFE_CALL(cudaMallocHost((void **)&shard, sizeof(shard_t)));
	//CUDA_SAFE_CALL(cudaMallocHost((void **)&shard->edges, MAX_NUM_EDGES_SHARD*sizeof(edge_t)));
	//CUDA_SAFE_CALL(cudaMallocManaged((void **)&shard, sizeof(shard_t)));
	//CUDA_SAFE_CALL(cudaMallocManaged((void **)&shard->edges, MAX_NUM_EDGES_SHARD*sizeof(edge_t)));

	edge_t * edges_dev;
	CUDA_SAFE_CALL(hipMalloc((void **)&shard, sizeof(shard_t)));
	CUDA_SAFE_CALL(hipMalloc((void **)&edges_dev, MAX_NUM_EDGES_SHARD*sizeof(edge_t)));

	// It will contain the visited status of each vertex
	vertex_t *vertices;
	//CUDA_SAFE_CALL(cudaMallocHost((void **)&vertices, num_vertices*sizeof(vertex_t)));
	vertex_t *vertices_host = (vertex_t *) malloc(num_vertices*sizeof(vertex_t));
	CUDA_SAFE_CALL(hipMalloc((void **)&vertices, num_vertices*sizeof(vertex_t)));

	init<<<((num_vertices+MAX_THREADS_PER_BLOCK-1)/MAX_THREADS_PER_BLOCK),MAX_THREADS_PER_BLOCK>>> (vertices, 0, num_vertices);

	hipEvent_t start,end;
	float diff;
	double time = 0;

	CUDA_SAFE_CALL(hipEventCreate(&start));
	CUDA_SAFE_CALL(hipEventCreate(&end));
	
	printf("Begin kernel\n");

	bool stop;
	k=0;
	do
	{
		stop = false;
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_over), &stop, sizeof(bool),0, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipDeviceSynchronize());

		for(i=0; i<num_intervals; i++)
		{
			//Load the data of shard_host[i] into shard (pinned memory)
			/*shard->E = shard_host[i].E;
			shard->Vstart = shard_host[i].Vstart;
			shard->Vend = shard_host[i].Vend;
			for (j=0; j<shard_host[i].E; j++)
			{
				shard->edges[j] = shard_host[i].edges[j];
				j++;
			}*/
			CUDA_SAFE_CALL(hipMemcpy(shard, &shard_host[i], sizeof(shard_t),hipMemcpyHostToDevice));
			CUDA_SAFE_CALL(hipMemcpy(edges_dev, shard_host[i].edges, shard_host[i].E*sizeof(edge_t),hipMemcpyHostToDevice));
			CUDA_SAFE_CALL(hipMemcpy(&(shard->edges), &edges_dev, sizeof(edge_t *),hipMemcpyHostToDevice));

			gettimeofday(&t1,NULL);

			scatter_bfs<<<grid, threads>>> (shard, vertices, k, num_vertices);

			CUDA_SAFE_CALL(hipDeviceSynchronize());
			gettimeofday(&t2,NULL);
			time += ((t2.tv_sec*1.0e3+t2.tv_usec*1.0e-3)-(t1.tv_sec*1.0e3+t1.tv_usec*1.0e-3));
		}
		/*for(i=0; i<num_intervals; i++)
		{
			//Load the data of shard_host[i] into shard (pinned memory)
			shard.E = shard_host[i].E;
			shard.Vstart = shard_host[i].Vstart;
			shard.Vend = shard_host[i].Vend;
			for (j=0; j<shard_host[i].E; j++)
			{
				shard.edges[j] = shard_host[i].edges[j];
				j++;
			}

			gettimeofday(&t1,NULL);

		 	gather_bfs<<<grid, threads>>> (shard, vertices, k, num_vertices);

			CUDA_SAFE_CALL(cudaDeviceSynchronize());
			gettimeofday(&t2,NULL);
			time += ((t2.tv_sec*1.0e3+t2.tv_usec*1.0e-3)-(t1.tv_sec*1.0e3+t1.tv_usec*1.0e-3))
		}*/

		CUDA_SAFE_CALL(hipMemcpyFromSymbol(&stop, HIP_SYMBOL(d_over), sizeof(bool),0, hipMemcpyDeviceToHost));
		k++;
	}while(stop);

	printf("Number of iterations : %d\n",k);
	CUDA_SAFE_CALL(hipMemcpy(vertices_host, vertices, num_vertices*sizeof(vertex_t), hipMemcpyDeviceToHost));
	/*for(int i = 0; i < num_vertices; i++)
	{
		printf("Vertex %d Distance %d\n",i,vertices_host[i].val);
	}*/
	printf("Time: %f ms\n",time);

	free(interval);
	for(i=0; i<num_intervals; i++)
	{
		free(shard_host[i].edges);
	}
	free(shard_host);
	free(vertices_host);
	//CUDA_SAFE_CALL(cudaFreeHost(vertices));
	//CUDA_SAFE_CALL(cudaFreeHost(shard->edges));
	//CUDA_SAFE_CALL(cudaFreeHost(shard));
	CUDA_SAFE_CALL(hipFree(vertices));
	CUDA_SAFE_CALL(hipFree(edges_dev));
	CUDA_SAFE_CALL(hipFree(shard));


	CUDA_SAFE_CALL(hipEventDestroy(start));
	CUDA_SAFE_CALL(hipEventDestroy(end));

	return 0;
}
