#include "hip/hip_runtime.h"
#include <stdio.h>

typedef unsigned long long int LONG;

void initarr(int *arr, LONG n)
{
	for(LONG i=0; i<n; i++)
		arr[i] = i;
}

void shuffle(int *arr, LONG n)
{
	initarr(arr,n);
	if (n > 1) 
	{
		LONG i;
		srand(time(NULL));
		for (i = 0; i < n - 1; i++) 
		{
			LONG j = i + rand() / (RAND_MAX / (n - i) + 1);
			LONG t = arr[j];
			arr[j] = arr[i];
			arr[i] = t;
		}
	}
}

double bandwidth(LONG n, double t)
{
	return ((double)n * (sizeof(double) + sizeof(int)) / t);
}

__global__
void kernel(double * A, int * T, LONG N)
{
	LONG i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i < N)
		A[T[i]] = (double) i / threadIdx.x;
}

int main(int argc, char *argv[])
{
	LONG N;
	hipEvent_t start,stop;
	float diff;
	double time, th2d, tunpin, tpin, tmgm;

	if(argc==1)
	{
		N = 100000000;
	}
	else if(argc==2)
	{
		N = atoi(argv[2]);
	}
	else
	{
		printf("./seq <N>");
		exit(-1);
	}

	hipSetDevice(0);
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for(N=1;N<=1000000000;N=N*10)
	{
	const LONG BLOCKSIZE = 1024;
	const LONG NUMBLOCKS = (N + BLOCKSIZE - 1) / BLOCKSIZE;

	/* Explicit Host to device and vice versa copies  */
	
	double* A_cpu; int* T_cpu;
	double* B_gpu; int* T_gpu;

	A_cpu = (double *) malloc(N * sizeof(double));
	hipMalloc((void **)&B_gpu, N * sizeof(double));

	T_cpu = (int *) malloc(N * sizeof(int));
	shuffle(T_cpu, N);
	hipMalloc((void **)&T_gpu, N * sizeof(int));
	
	hipEventRecord(start, 0);
	hipMemcpy((void *)B_gpu, (void *)A_cpu, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy((void *)T_gpu, (void *)T_cpu, N * sizeof(int), hipMemcpyHostToDevice);

	kernel<<<NUMBLOCKS, BLOCKSIZE>>>(B_gpu,T_gpu,N);
	hipDeviceSynchronize();	
	hipMemcpy((void *)A_cpu, (void *)B_gpu, N * sizeof(double), hipMemcpyDeviceToHost);
	for(LONG i = 0; i < N; i++)
		A_cpu[T_cpu[i]] += i;

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&diff,start,stop);
	time = diff * 1.0e-3;

	//printf("Explicit H2D & D2H bandwidth : %lf GB/s\n",bandwidth(N,time) * 1.0e-9);
	th2d = time;

	free(A_cpu); free(T_cpu);
	hipFree(B_gpu); hipFree(T_gpu);

	/* UVA unpinned memory  */
	
	double* C_cpu;
	double* D_gpu;

	C_cpu = (double *) malloc(N * sizeof(double));
	hipMalloc((void **)&D_gpu, N * sizeof(double));

	T_cpu = (int *) malloc(N * sizeof(int));
	shuffle(T_cpu, N);
	hipMalloc((void **)&T_gpu, N * sizeof(int));

	hipEventRecord(start, 0);
	hipMemcpy((void *)D_gpu, (void *)C_cpu, N * sizeof(double), hipMemcpyDefault);
	hipMemcpy((void *)T_gpu, (void *)T_cpu, N * sizeof(int), hipMemcpyDefault);
	kernel<<<NUMBLOCKS, BLOCKSIZE>>>(D_gpu,T_gpu,N);
	hipDeviceSynchronize();	
	hipMemcpy((void *)C_cpu, (void *)D_gpu, N * sizeof(double), hipMemcpyDefault);
	for(LONG i = 0; i < N; i++)
		C_cpu[T_cpu[i]] += i;

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&diff,start,stop);
	time = diff * 1.0e-3;

	//printf("UVA unpinned bandwidth : %lf GB/s\n",bandwidth(N,time) * 1.0e-9);
	tunpin = time;

	free(C_cpu);
	hipFree(D_gpu);

	/* UVA pinned memory  */
	
	double* E_cpu;

	hipHostAlloc ((void **)&E_cpu, N * sizeof(double), hipHostMallocMapped /*| hipHostMallocPortable*/);

	hipHostAlloc ((void **)&T_cpu, N * sizeof(int), hipHostMallocMapped /*| hipHostMallocPortable*/);
	shuffle(T_cpu, N);
	
	hipEventRecord(start, 0);
	kernel<<<NUMBLOCKS, BLOCKSIZE>>>(E_cpu,T_cpu,N);
	hipDeviceSynchronize();
	for(LONG i = 0; i < N; i++)
		E_cpu[T_cpu[i]] += i;

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&diff,start,stop);
	time = diff * 1.0e-3;

	//printf("UVA pinned bandwidth : %lf GB/s\n",bandwidth(N,time) * 1.0e-9);
	tpin = time;

	hipHostFree(E_cpu);
	hipHostFree(T_cpu);

	/* Unified memory  */
	
	double* F_cpu;

	hipMallocManaged((void **)&F_cpu, N * sizeof(double));

	hipMallocManaged((void **)&T_cpu, N * sizeof(int));
	shuffle(T_cpu, N);
	
	hipEventRecord(start, 0);

	kernel<<<NUMBLOCKS, BLOCKSIZE>>>(F_cpu,T_cpu,N);
	hipDeviceSynchronize();
	for(LONG i = 0; i < N; i++)
		F_cpu[T_cpu[i]] += i;

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&diff,start,stop);
	time = diff * 1.0e-3;

	//printf("Unified memory bandwidth : %lf GB/s\n",bandwidth(N,time) * 1.0e-9);
	tmgm = time;
	
	hipFree(F_cpu);
	hipFree(T_cpu);

	printf("%llu %lf %lf %lf %lf\n",N, th2d, tunpin, tpin, tmgm);
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
