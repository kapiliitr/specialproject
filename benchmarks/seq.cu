
#include <hip/hip_runtime.h>
#include <stdio.h>

typedef unsigned long long int LONG;

double bandwidth(LONG n, double t)
{
	return ((double)n * sizeof(double) / t);
}

__global__
void kernel(double * A, LONG N)
{
	LONG i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i < N)
		A[i] = (double) i / threadIdx.x;
}

int main(int argc, char *argv[])
{
	LONG N;
	hipEvent_t start,stop;
	float diff;
	double time, th2d, tunpin, tpin, tmgm;

	if(argc==1)
	{
		N = 100000000;
	}
	else if(argc==2)
	{
		N = atoi(argv[2]);
	}
	else
	{
		printf("./seq <N>");
		exit(-1);
	}
	
	hipSetDevice(0);
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for(N=1;N<=1000000000;N=N*10)
	{
	const LONG BLOCKSIZE = 1024;
	const LONG NUMBLOCKS = (N + BLOCKSIZE - 1) / BLOCKSIZE;

	/* Explicit Host to device and vice versa copies  */
	
	double* A_cpu;
	double* B_gpu;

	A_cpu = (double *) malloc(N * sizeof(double));
	hipMalloc((void **)&B_gpu, N * sizeof(double));
	hipEventRecord(start, 0);
	hipMemcpy((void *)B_gpu, (void *)A_cpu, N * sizeof(double), hipMemcpyHostToDevice);
	kernel<<<NUMBLOCKS, BLOCKSIZE>>>(B_gpu,N);
	hipDeviceSynchronize();
	hipMemcpy((void *)A_cpu, (void *)B_gpu, N * sizeof(double), hipMemcpyDeviceToHost);
	for(LONG i = 0; i < N; i++)
		A_cpu[i] += i;

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&diff,start,stop);
	time = diff * 1.0e-3;
	//printf("Explicit H2D & D2H bandwidth : %lf GB/s\tTime : %lf s\n",bandwidth(N,time) * 1.0e-9,time);
	th2d = time;

	free(A_cpu);
	hipFree(B_gpu);

	/* UVA unpinned memory  */
	
	double* C_cpu;
	double* D_gpu;

	C_cpu = (double *) malloc(N * sizeof(double));
	hipMalloc((void **)&D_gpu, N * sizeof(double));
	hipEventRecord(start, 0);
	hipMemcpy((void *)D_gpu, (void *)C_cpu, N * sizeof(double), hipMemcpyDefault);
	kernel<<<NUMBLOCKS, BLOCKSIZE>>>(D_gpu,N);
	hipDeviceSynchronize();
	hipMemcpy((void *)C_cpu, (void *)D_gpu, N * sizeof(double), hipMemcpyDefault);

	for(LONG i = 0; i < N; i++)
		C_cpu[i] += i;

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&diff,start,stop);

	time = diff * 1.0e-3;
	//printf("UVA unpinned bandwidth : %lf GB/s\tTime : %lf s\n",bandwidth(N,time) * 1.0e-9,time);
	tunpin = time;

	free(C_cpu);
	hipFree(D_gpu);

	/* UVA pinned memory  */
	
	double* E_cpu;

	hipHostAlloc ((void **)&E_cpu, N * sizeof(double), hipHostMallocMapped /*| cudaHostAllocPortable*/);
	hipEventRecord(start, 0);
	kernel<<<NUMBLOCKS, BLOCKSIZE>>>(E_cpu,N);
	hipDeviceSynchronize();

	for(LONG i = 0; i < N; i++)
		E_cpu[i] += i;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&diff,start,stop);

	time = diff * 1.0e-3;
	//printf("UVA pinned bandwidth : %lf GB/s\tTime : %lf s\n",bandwidth(N,time) * 1.0e-9,time);
	tpin = time;

	hipHostFree(E_cpu);

	/* Unified memory  */
	
#if 1
	double* F_cpu;

	hipMallocManaged((void **)&F_cpu, N * sizeof(double));
	hipEventRecord(start, 0);
	kernel<<<NUMBLOCKS, BLOCKSIZE>>>(F_cpu,N);
	hipDeviceSynchronize();

	for(LONG i = 0; i < N; i++)
		F_cpu[i] += i;

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&diff,start,stop);
	time = diff * 1.0e-3;
	//printf("Unified memory bandwidth : %lf GB/s\tTime : %lf s\n",bandwidth(N,time) * 1.0e-9,time);
	tmgm = time;
	
	hipFree(F_cpu);
#endif
	printf("%llu %lf %lf %lf %lf\n",N, th2d, tunpin, tpin, tmgm);
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
