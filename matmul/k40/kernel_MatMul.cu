
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdio>
#include <sys/time.h>

using namespace std;

#define CUDA_SAFE_CALL( err ) (safe_call(err, __LINE__))
#define BLOCK_SIZE 32
#define ERROR 1.0e-9

typedef unsigned long long int LONG;

void printArr(double *A, LONG N)
{
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
			cout << A[i*N+j] << " ";
		cout << endl;
	}
}

void safe_call(hipError_t ret, int line)
{
	if(ret!=hipSuccess)
	{
		cout << "Error at line " << line << " : " << hipGetErrorString(ret) << endl;
		exit(-1);
	}
}

void printMat(double *A, LONG N)
{
	LONG i,j;
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
			cout << A[i*N+j] << " ";
		cout<<endl;
	}
}

__global__ void gpuMM(double *A, double *B, double *C, LONG N)
{
	// Matrix multiplication for NxN matrices C=A*B
	// Each thread computes a single element of C
	LONG row = threadIdx.y;
	LONG col = blockIdx.x*blockDim.x + threadIdx.x;

	double sum = 0.f;
	for (LONG n = 0; n < N; n++)
	    sum += A[row*N+n]*B[n*N+col];

	C[row*N+col] = sum;
}

int main(int argc, char *argv[])
{
	struct timeval t1,t2, tp;
	double tt, gflops;

	// Perform matrix multiplication C = A*B
	// where A, B and C are NxN matrices
	// Restricted to matrices where N = K*BLOCK_SIZE;
	LONG N,K,S;
	cin >> K >> S;
	N = K*BLOCK_SIZE;
	if(N%S)
	{
		cout << S << " should be divisible by " << N << endl;
		return 0;
	}

	CUDA_SAFE_CALL(hipSetDevice(0));

	cout << "Executing Matrix Multiplcation" << endl;
	cout << "Matrix size: " << N << "x" << N << endl;

	// Allocate memory on the host
	double *hA,*hB,*hC;
	hA = new double[N*N];
	hB = new double[N*N];
	hC = new double[N*N];

	// Initialize matrices on the host
	srand(time(NULL));
	for (LONG j=0; j<N; j++){
	    for (LONG i=0; i<N; i++){
	    	hA[j*N+i] = drand48();
		hB[j*N+i] = drand48();
	    }
	}

	// Allocate memory on the device
	LONG size = N*N*sizeof(double);	// Size of the memory in bytes
	double *dA,*dB,*dC,*dAT,*dCT;
	
	// Allocate memory to store the GPU answer on the host
	double *C;
	C = new double[N*N];	

	CUDA_SAFE_CALL(hipHostMalloc(&dB,size, hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dA,(S*size/N), hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dC,(S*size/N), hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dAT,(S*size/N), hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dCT,(S*size/N), hipHostMallocDefault));
	
	dim3 threadBlock(BLOCK_SIZE,S);
	dim3 grid(K);

	hipStream_t * str = (hipStream_t *) malloc((N/S) * sizeof(hipStream_t));
	hipEvent_t * evt = (hipEvent_t *) malloc((N/S) * sizeof(hipEvent_t));
	for(int i = 0; i < (N/S); i++)
	{
	        CUDA_SAFE_CALL(hipStreamCreate(&(str[i])));
		CUDA_SAFE_CALL(hipEventCreate(&(evt[i])));
	}

	gettimeofday(&t1,0);

	// Copy matrices from the host to device
	CUDA_SAFE_CALL(hipMemcpyAsync(dB,hB,size,hipMemcpyHostToDevice,str[0]));

	CUDA_SAFE_CALL(hipMemcpyAsync(dA,hA,S*(size/N),hipMemcpyHostToDevice,str[0]));
	gpuMM<<<grid,threadBlock,0,str[0]>>>(dA,dB,dC,N);
	CUDA_SAFE_CALL(hipEventRecord(evt[0],str[0]));
	for(LONG i=1; i< (N/S); i++){
		if(i%2 == 0)
		{
			//Wait for previous stream to finish executing the kernel
			CUDA_SAFE_CALL(hipStreamWaitEvent(str[i],evt[i-2],0));

			// Prefetch the next set of rows
			CUDA_SAFE_CALL(hipMemcpyAsync(dA,hA+i*N*S,(S*size/N),hipMemcpyHostToDevice,str[i]));

			CUDA_SAFE_CALL(hipStreamSynchronize(str[i-2]));

			//Execute the matrix multiplication kernel
			gpuMM<<<grid,threadBlock,0,str[i]>>>(dA,dB,dC,N);
			CUDA_SAFE_CALL(hipEventRecord(evt[i],str[i]));

			// Now copy the GPU result back to CPU
			CUDA_SAFE_CALL(hipMemcpyAsync(C+(i-1)*N*S,dCT,(S*size/N),hipMemcpyDeviceToHost,str[i-1]));
		}
		else
		{
			//Wait for previous stream to finish executing the kernel
			if(i>1)
				CUDA_SAFE_CALL(hipStreamWaitEvent(str[i],evt[i-2],0));

			// Prefetch the next set of rows
			CUDA_SAFE_CALL(hipMemcpyAsync(dAT,hA+i*N*S,(S*size/N),hipMemcpyHostToDevice,str[i]));

			if(i>1)
				CUDA_SAFE_CALL(hipStreamSynchronize(str[i-2]));

			//Execute the matrix multiplication kernel
			gpuMM<<<grid,threadBlock,0,str[i]>>>(dAT,dB,dCT,N);
			CUDA_SAFE_CALL(hipEventRecord(evt[i],str[i]));

			// Now copy the GPU result back to CPU
			CUDA_SAFE_CALL(hipMemcpyAsync(C+(i-1)*N*S,dC,(S*size/N),hipMemcpyDeviceToHost,str[i-1]));
		}
	}
	CUDA_SAFE_CALL(hipStreamSynchronize(str[(N/S)-1]));
	if(((N/S)-1)%2 == 0)
		CUDA_SAFE_CALL(hipMemcpyAsync(C+((N/S)-1)*N*S,dC,(S*size/N),hipMemcpyDeviceToHost,str[(N/S)-1]));
	else
		CUDA_SAFE_CALL(hipMemcpyAsync(C+((N/S)-1)*N*S,dCT,(S*size/N),hipMemcpyDeviceToHost,str[(N/S)-1]));
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	gettimeofday(&t2,0);
	timersub(&t2,&t1,&tp);

	tt = (double) tp.tv_sec + ((double) tp.tv_usec/1.0e6);
	gflops = ( 1.0e-9 * 2.0 * N * N * N ) / tt; 
	cout << "Prefetch : " << gflops << endl; 

	for(int i = 0; i < (N/S); i++)
	{
		CUDA_SAFE_CALL(hipStreamDestroy(str[i]));
		CUDA_SAFE_CALL(hipEventDestroy(evt[i]));
	}

#if 0
	// Now do the matrix multiplication on the CPU
	double sum;
	for (LONG row=0; row<N; row++){
		for (LONG col=0; col<N; col++){
			sum = 0.f;
			for (LONG n=0; n<N; n++){
				sum += hA[row*N+n]*hB[n*N+col];
			}
			hC[row*N+col] = sum;
		}
	}

	// Check the result and make sure it is correct
	for (LONG row=0; row<N; row++){
		for (LONG col=0; col<N; col++){
			if ( fabs(C[row*N+col] - hC[row*N+col]) > ERROR ){
				cout << "Wrong answer!" << row << " " << col << endl;
				row = col = N;
			}
		}
	}
	
	printArr(C,N);
	cout<<endl;
	printArr(hC,N);

#endif

	CUDA_SAFE_CALL(hipHostFree(dB));
	CUDA_SAFE_CALL(hipHostFree(dA));
	CUDA_SAFE_CALL(hipHostFree(dC));
	CUDA_SAFE_CALL(hipHostFree(dAT));

	cout << "Finished." << endl;

	return 0;
}
