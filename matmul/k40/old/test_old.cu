
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdio>
#include <sys/time.h>

using namespace std;

#define CUDA_SAFE_CALL( err ) (safe_call(err, __LINE__))
#define BLOCK_SIZE 32
#define ERROR 1.0e-9

typedef unsigned long long int LONG;

void safe_call(hipError_t ret, int line)
{
	if(ret!=hipSuccess)
	{
		cout << "Error at line " << line << " : " << hipGetErrorString(ret) << endl;
		exit(-1);
	}
}

void printMat(double *A, LONG N)
{
	LONG i,j;
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
			cout << A[i*N+j] << " ";
		cout<<endl;
	}
}

__global__ void gpuMM(double *A, double *B, double *C, LONG N)
{
	// Matrix multiplication for NxN matrices C=A*B
	// Each thread computes a single element of C
	LONG row = threadIdx.y;
	LONG col = blockIdx.x*blockDim.x + threadIdx.x;

	double sum = 0.f;
	for (LONG n = 0; n < N; n++)
	    sum += A[row*N+n]*B[n*N+col];

	C[row*N+col] = sum;
}

__global__ void gpuMM_um(double *A, double *B, double *C, LONG N)
{
	// Matrix multiplication for NxN matrices C=A*B
	// Each thread computes a single element of C
	LONG row = blockIdx.y*blockDim.y + threadIdx.y;
	LONG col = blockIdx.x*blockDim.x + threadIdx.x;

	double sum = 0.f;
	for (LONG n = 0; n < N; ++n)
	    sum += A[row*N+n]*B[n*N+col];

	C[row*N+col] = sum;
}

int main(int argc, char *argv[])
{
	struct timeval t1,t2, tnp, tp;
	double tt, gflops;

	// Perform matrix multiplication C = A*B
	// where A, B and C are NxN matrices
	// Restricted to matrices where N = K*BLOCK_SIZE;
	LONG N,K;
	cin >> K;
	N = K*BLOCK_SIZE;

	CUDA_SAFE_CALL(hipSetDevice(0));

	cout << "Executing Matrix Multiplcation" << endl;
	cout << "Matrix size: " << N << "x" << N << endl;

	// Allocate memory on the host
	double *hA,*hB,*hC;
	hA = new double[N*N];
	hB = new double[N*N];
	hC = new double[N*N];

	// Initialize matrices on the host
	srand(time(NULL));
	for (LONG j=0; j<N; j++){
	    for (LONG i=0; i<N; i++){
	    	hA[j*N+i] = drand48();
		hB[j*N+i] = drand48();
	    }
	}

	// Allocate memory on the device
	LONG size = N*N*sizeof(double);	// Size of the memory in bytes
	
	// Allocate memory to store the GPU answer on the host
	double *C;
	C = new double[N*N];	

	dim3 threadBlock(BLOCK_SIZE,K);
	dim3 grid(K);

	double *dA,*dB,*dC,*dAT,*dCT,*dTemp;

	/* With prefetching begins  */

	CUDA_SAFE_CALL(hipHostMalloc(&dB,size, hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dA,(K*size/N), hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dC,(K*size/N), hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dAT,(K*size/N), hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dCT,(K*size/N), hipHostMallocDefault));
	
	hipStream_t s1,s2,s3;
	CUDA_SAFE_CALL(hipStreamCreate(&s1));
	CUDA_SAFE_CALL(hipStreamCreate(&s2));
	CUDA_SAFE_CALL(hipStreamCreate(&s3));

	gettimeofday(&t1,0);

	// Copy matrices from the host to device
	CUDA_SAFE_CALL(hipMemcpyAsync(dB,hB,size,hipMemcpyHostToDevice,s1));

	CUDA_SAFE_CALL(hipMemcpyAsync(dA,hA,K*(size/N),hipMemcpyHostToDevice,s1));
	gpuMM<<<grid,threadBlock,0,s1>>>(dA,dB,dC,N);
	for(LONG i=1; i< (N/K); i++){
		// Prefetch the next set of rows
		CUDA_SAFE_CALL(hipMemcpyAsync(dAT,hA+i*N*K,(K*size/N),hipMemcpyHostToDevice,s2));

		CUDA_SAFE_CALL(hipDeviceSynchronize());
		
		//Swap pointers
		dTemp = dAT;
		dAT = dA;
		dA = dTemp;

		dTemp = dCT;
		dCT = dC;
		dC = dTemp;

		//Execute the matrix multiplication kernel
		gpuMM<<<grid,threadBlock,0,s1>>>(dA,dB,dC,N);

		// Now copy the GPU result back to CPU
		CUDA_SAFE_CALL(hipMemcpyAsync(C+(i-1)*N*K,dCT,(K*size/N),hipMemcpyDeviceToHost,s3));
	}
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUDA_SAFE_CALL(hipMemcpyAsync(C+((N/K)-1)*N*K,dC,(K*size/N),hipMemcpyDeviceToHost,s3));

	gettimeofday(&t2,0);
	timersub(&t2,&t1,&tp);

	tt = (double) tp.tv_sec + ((double) tp.tv_usec/1.0e6);
	gflops = ( 1.0e-9 * 2.0 * N * N * N ) / tt; 
	cout << "Prefetch : " << gflops << endl; 

	CUDA_SAFE_CALL(hipStreamDestroy(s1));
	CUDA_SAFE_CALL(hipStreamDestroy(s2));
	CUDA_SAFE_CALL(hipStreamDestroy(s3));

	CUDA_SAFE_CALL(hipHostFree(dB));
	CUDA_SAFE_CALL(hipHostFree(dA));
	CUDA_SAFE_CALL(hipHostFree(dC));
	CUDA_SAFE_CALL(hipHostFree(dAT));
	CUDA_SAFE_CALL(hipHostFree(dCT));

	/* Without prefetching begins  */
	
	CUDA_SAFE_CALL(hipMalloc(&dB,size));
	CUDA_SAFE_CALL(hipMalloc(&dA,(K*size/N)));
	CUDA_SAFE_CALL(hipMalloc(&dC,(K*size/N)));

	gettimeofday(&t1,0);
	CUDA_SAFE_CALL(hipMemcpy(dB,hB,size,hipMemcpyHostToDevice));
	for(LONG i=0; i< (N/K); i++){
		//cout << "Iteration " << i << endl;
	
		CUDA_SAFE_CALL(hipMemcpy(dA,hA+i*N*K,(K*size/N),hipMemcpyHostToDevice));
	
		//Execute the matrix multiplication kernel	
		gpuMM<<<grid,threadBlock>>>(dA,dB,dC,N);
	
		// Now copy the GPU result back to CPU
		CUDA_SAFE_CALL(hipMemcpy(C+i*N*K,dC,(K*size/N),hipMemcpyDeviceToHost));

	}
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	gettimeofday(&t2,0);
	timersub(&t2,&t1,&tnp);

	tt = (double) tnp.tv_sec + ((double) tnp.tv_usec/1.0e6);
	gflops = ( 1.0e-9 * 2.0 * N * N * N ) / tt; 
	cout << "Without Prefetch : " << gflops << endl; 

	CUDA_SAFE_CALL(hipFree(dB));
	CUDA_SAFE_CALL(hipFree(dA));
	CUDA_SAFE_CALL(hipFree(dC));

	/* With Managed memory begins  */

	CUDA_SAFE_CALL(hipMallocManaged(&dA,size));
	CUDA_SAFE_CALL(hipMallocManaged(&dB,size));
	CUDA_SAFE_CALL(hipMallocManaged(&dC,size));

	dim3 threadBlock_um(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid_um(K,K);

	// Initialize matrices
	for (LONG j=0; j<N; j++){
	    for (LONG i=0; i<N; i++){
	    	dA[j*N+i] = 2.f*(j+i);
		dB[j*N+i] = 1.f*(j-i);
	    }
	}
	
	gettimeofday(&t1,0);

	gpuMM_um<<<grid_um,threadBlock_um>>>(dA,dB,dC,N);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	
	gettimeofday(&t2,0);
	timersub(&t2,&t1,&tp);

	tt = (double) tp.tv_sec + ((double) tp.tv_usec/1.0e6);
	gflops = ( 1.0e-9 * 2.0 * N * N * N ) / tt; 
	cout << "Managed : " << gflops << endl; 

	CUDA_SAFE_CALL(hipFree(dA));
	CUDA_SAFE_CALL(hipFree(dB));
	CUDA_SAFE_CALL(hipFree(dC));

	delete [] hA;
	delete [] hB;
	delete [] hC;
	delete [] C;

	cout << "Finished." << endl;

	return 0;
}
