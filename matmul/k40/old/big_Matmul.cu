
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdio>
#include <sys/time.h>

using namespace std;

#define CUDA_SAFE_CALL( err ) (safe_call(err, __LINE__))
#define BLOCK_SIZE 32
#define ERROR 1.0e-9

typedef unsigned long long int LONG;

void safe_call(hipError_t ret, int line)
{
	if(ret!=hipSuccess)
	{
		cout << "Error at line " << line << " : " << hipGetErrorString(ret) << endl;
		exit(-1);
	}
}

void printMat(double *A, LONG N)
{
	LONG i,j;
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
			cout << A[i*N+j] << " ";
		cout<<endl;
	}
}

__global__ void gpuMM(double *A, double *B, double *C, LONG N)
{
	// Matrix multiplication for NxN matrices C=A*B
	// Each thread computes a single element of C
	LONG row = threadIdx.y;
	LONG col = blockIdx.x*blockDim.x + threadIdx.x;

	double sum = 0.f;
	for (LONG n = 0; n < N; n++)
	    sum += A[row*N+n]*B[n*N+col];

	C[row*N+col] = sum;
}

int main(int argc, char *argv[])
{
	struct timeval t1,t2, tnp;
	double tt, gflops;

	// Perform matrix multiplication C = A*B
	// where A, B and C are NxN matrices
	// Restricted to matrices where N = K*BLOCK_SIZE;
	LONG N,K;
	cin >> K;
	N = K*BLOCK_SIZE;

	CUDA_SAFE_CALL(hipSetDevice(0));

	cout << "Executing Matrix Multiplcation" << endl;
	cout << "Matrix size: " << N << "x" << N << endl;

	// Allocate memory on the host
	double *hA,*hB,*hC;
	hA = new double[N*N];
	hB = new double[N*N];
	hC = new double[N*N];

	// Initialize matrices on the host
	srand(time(NULL));
	for (LONG j=0; j<N; j++){
	    for (LONG i=0; i<N; i++){
	    	hA[j*N+i] = drand48();
		hB[j*N+i] = drand48();
	    }
	}

	// Allocate memory on the device
	LONG size = N*N*sizeof(double);	// Size of the memory in bytes
	double *dA,*dB,*dC;
	
	// Allocate memory to store the GPU answer on the host
	double *C;
	C = new double[N*N];	

	CUDA_SAFE_CALL(hipMalloc(&dB,size));
	CUDA_SAFE_CALL(hipMalloc(&dA,(K*size/N)));
	CUDA_SAFE_CALL(hipMalloc(&dC,(K*size/N)));
	
	dim3 threadBlock(BLOCK_SIZE,K);
	dim3 grid(K);

	gettimeofday(&t1,0);
	CUDA_SAFE_CALL(hipMemcpy(dB,hB,size,hipMemcpyHostToDevice));
	for(LONG i=0; i< (N/K); i++){
		//cout << "Iteration " << i << endl;
	
		CUDA_SAFE_CALL(hipMemcpy(dA,hA+i*N*K,(K*size/N),hipMemcpyHostToDevice));
	
		//Execute the matrix multiplication kernel	
		gpuMM<<<grid,threadBlock>>>(dA,dB,dC,N);
	
		// Now copy the GPU result back to CPU
		CUDA_SAFE_CALL(hipMemcpy(C+i*N*K,dC,(K*size/N),hipMemcpyDeviceToHost));

	}
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	gettimeofday(&t2,0);
	timersub(&t2,&t1,&tnp);

	tt = (double) tnp.tv_sec + ((double) tnp.tv_usec/1.0e6);
	gflops = ( 1.0e-9 * 2.0 * N * N * N ) / tt; 
	cout << "Without Prefetch : " << gflops << endl; 

	cout << "Device operations done." << endl;

	CUDA_SAFE_CALL(hipFree(dB));
	CUDA_SAFE_CALL(hipFree(dA));
	CUDA_SAFE_CALL(hipFree(dC));

	cout << "Finished." << endl;
	
	return 0;
}
