
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdio>
#include <sys/time.h>

using namespace std;

#define CUDA_SAFE_CALL( err ) (safe_call(err, __LINE__))
#define BLOCK_SIZE 32
#define ERROR 1.0e-9

typedef unsigned long long int LONG;

void safe_call(hipError_t ret, int line)
{
	if(ret!=hipSuccess)
	{
		cout << "Error at line " << line << " : " << hipGetErrorString(ret) << endl;
		exit(-1);
	}
}

void printMat(double *A, LONG N)
{
	LONG i,j;
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
			cout << A[i*N+j] << " ";
		cout<<endl;
	}
}

__global__ void gpuMM(double *A, double *B, double *C, LONG N)
{
	// Matrix multiplication for NxN matrices C=A*B
	// Each thread computes a single element of C
	LONG row = threadIdx.y;
	LONG col = blockIdx.x*blockDim.x + threadIdx.x;

	double sum = 0.f;
	for (LONG n = 0; n < N; n++)
	{
	    sum += A[row*N+n]*B[n*N+col];
	}

	C[row*N+col] = sum;
}

__global__ void gpuMM_um(double *A, double *B, double *C, LONG N)
{
	// Matrix multiplication for NxN matrices C=A*B
	// Each thread computes a single element of C
	LONG row = blockIdx.y*blockDim.y + threadIdx.y;
	LONG col = blockIdx.x*blockDim.x + threadIdx.x;

	double sum = 0.f;
	for (LONG n = 0; n < N; ++n)
	    sum += A[row*N+n]*B[n*N+col];

	C[row*N+col] = sum;
}

int main(int argc, char *argv[])
{
	struct timeval t1,t2, tnp, tp;
	double tt, gflops, tpre, tprenk, tnprepin, tnpre, tmgm;

	/*if(argc != 3)
	{
		cout << "./test K S" << endl;
		return 0;
	}*/

	// Perform matrix multiplication C = A*B
	// where A, B and C are NxN matrices
	// Restricted to matrices where N = K*BLOCK_SIZE;
	LONG N,K,S;
	//K = atoi(argv[1]);
	//S = atoi(argv[2]);
	S = atoi(argv[1]);
	for(K=50; K<=400; K+=50)
	{
	N = K*BLOCK_SIZE;
	if(N%S)
	{
		cout << S << " should be divisible by " << N << endl;
		return 0;
	}

	CUDA_SAFE_CALL(hipSetDevice(0));

	//cout << "Executing Matrix Multiplcation" << endl;
	//cout << "Matrix size: " << N << "x" << N << endl;

	// Allocate memory on the host
	double *hA,*hB,*hC;
	hA = new double[N*N];
	hB = new double[N*N];
	hC = new double[N*N];

	// Initialize matrices on the host
	srand(time(NULL));
	for (LONG j=0; j<N; j++){
	    for (LONG i=0; i<N; i++){
	    	hA[j*N+i] = drand48();
		hB[j*N+i] = drand48();
	    }
	}

	// Allocate memory on the device
	LONG size = N*N*sizeof(double);	// Size of the memory in bytes
	
	// Allocate memory to store the GPU answer on the host
	double *C;
	C = new double[N*N];	

	dim3 threadBlock(BLOCK_SIZE,S);
	dim3 grid(K);

	double *dA,*dB,*dC,*dAT,*dCT;
	hipStream_t * str;
	hipEvent_t * evt;

#if 1
	/* With prefetching and launching kernel as well begins  */

	CUDA_SAFE_CALL(hipHostMalloc(&dB,size, hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dA,(S*size/N), hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dC,(S*size/N), hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dAT,(S*size/N), hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dCT,(S*size/N), hipHostMallocDefault));
	
	str = (hipStream_t *) malloc((N/S) * sizeof(hipStream_t));
	evt = (hipEvent_t *) malloc((N/S) * sizeof(hipEvent_t));
	for(int i = 0; i < (N/S); i++)
	{
	        CUDA_SAFE_CALL(hipStreamCreate(&(str[i])));
		CUDA_SAFE_CALL(hipEventCreate(&(evt[i])));
	}

	gettimeofday(&t1,0);

	// Copy matrices from the host to device
	CUDA_SAFE_CALL(hipMemcpyAsync(dB,hB,size,hipMemcpyHostToDevice,str[0]));
	for(LONG i=0; i< (N/S); i++){
		if(i%2 == 0)
		{
			if(i>0)
				CUDA_SAFE_CALL(hipStreamWaitEvent(str[i],evt[i-2],0));
			CUDA_SAFE_CALL(hipMemcpyAsync(dA,hA+i*N*S,(S*size/N),hipMemcpyHostToDevice,str[i]));
			if(i>0)
				CUDA_SAFE_CALL(hipStreamSynchronize(str[i-2]));
			gpuMM<<<grid,threadBlock,0,str[i]>>>(dA,dB,dC,N);
			CUDA_SAFE_CALL(hipEventRecord(evt[i],str[i]));
			CUDA_SAFE_CALL(hipMemcpyAsync(C+i*N*S,dC,(S*size/N),hipMemcpyDeviceToHost,str[i]));
		}
		else
		{
			if(i>1)
				CUDA_SAFE_CALL(hipStreamWaitEvent(str[i],evt[i-2],0));
			CUDA_SAFE_CALL(hipMemcpyAsync(dAT,hA+i*N*S,(S*size/N),hipMemcpyHostToDevice,str[i]));
			if(i>1)
				CUDA_SAFE_CALL(hipStreamSynchronize(str[i-2]));
			gpuMM<<<grid,threadBlock,0,str[i]>>>(dAT,dB,dCT,N);
			CUDA_SAFE_CALL(hipEventRecord(evt[i],str[i]));
			CUDA_SAFE_CALL(hipMemcpyAsync(C+i*N*S,dCT,(S*size/N),hipMemcpyDeviceToHost,str[i]));
		}
	}
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	gettimeofday(&t2,0);
	timersub(&t2,&t1,&tp);

	tt = (double) tp.tv_sec + ((double) tp.tv_usec/1.0e6);
	gflops = ( 1.0e-9 * 2.0 * N * N * N ) / tt; 
	//cout << "Prefetch : " << gflops << " Time : " << tt << " s"  << endl; 
	tpre = tt;

	for(int i = 0; i < (N/S); i++)
	{
		CUDA_SAFE_CALL(hipStreamDestroy(str[i]));
		CUDA_SAFE_CALL(hipEventDestroy(evt[i]));
	}

	CUDA_SAFE_CALL(hipHostFree(dB));
	CUDA_SAFE_CALL(hipHostFree(dA));
	CUDA_SAFE_CALL(hipHostFree(dC));
	CUDA_SAFE_CALL(hipHostFree(dAT));
	CUDA_SAFE_CALL(hipHostFree(dCT));

	/* With prefetching only begins  */

	CUDA_SAFE_CALL(hipHostMalloc(&dB,size, hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dA,(S*size/N), hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dC,(S*size/N), hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dAT,(S*size/N), hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dCT,(S*size/N), hipHostMallocDefault));
	
	str = (hipStream_t *) malloc((N/S) * sizeof(hipStream_t));
	evt = (hipEvent_t *) malloc((N/S) * sizeof(hipEvent_t));
	for(int i = 0; i < (N/S); i++)
	{
	        CUDA_SAFE_CALL(hipStreamCreate(&(str[i])));
		CUDA_SAFE_CALL(hipEventCreate(&(evt[i])));
	}

	gettimeofday(&t1,0);

	// Copy matrices from the host to device
	CUDA_SAFE_CALL(hipMemcpyAsync(dB,hB,size,hipMemcpyHostToDevice,str[0]));

	for(LONG i=0; i< (N/S); i++){
		if(i%2 == 0)
		{
			CUDA_SAFE_CALL(hipMemcpyAsync(dA,hA+i*N*S,(S*size/N),hipMemcpyHostToDevice,str[i]));
			if(i>0)
				CUDA_SAFE_CALL(hipStreamSynchronize(str[i-1]));
			gpuMM<<<grid,threadBlock,0,str[i]>>>(dA,dB,dC,N);
			CUDA_SAFE_CALL(hipMemcpyAsync(C+i*N*S,dC,(S*size/N),hipMemcpyDeviceToHost,str[i]));
		}
		else
		{
			CUDA_SAFE_CALL(hipMemcpyAsync(dAT,hA+i*N*S,(S*size/N),hipMemcpyHostToDevice,str[i]));
			CUDA_SAFE_CALL(hipStreamSynchronize(str[i-1]));
			gpuMM<<<grid,threadBlock,0,str[i]>>>(dAT,dB,dCT,N);
			CUDA_SAFE_CALL(hipMemcpyAsync(C+i*N*S,dCT,(S*size/N),hipMemcpyDeviceToHost,str[i]));
		}
	}
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	gettimeofday(&t2,0);
	timersub(&t2,&t1,&tp);

/*	// Check the result and make sure it is correct
	for (LONG row=0; row<N; row++){
		for (LONG col=0; col<N; col++){
			if ( fabs(C[row*N+col] - hC[row*N+col]) > ERROR ){
				cout << "Wrong answer!" << endl;
				row = col = N;
			}
		}
	}	*/

	tt = (double) tp.tv_sec + ((double) tp.tv_usec/1.0e6);
	gflops = ( 1.0e-9 * 2.0 * N * N * N ) / tt; 
	//cout << "Prefetch without kernel launch : " << gflops << " Time : " << tt << " s"  << endl; 
	tprenk = tt;

	for(int i = 0; i < (N/S); i++)
	{
		CUDA_SAFE_CALL(hipStreamDestroy(str[i]));
		CUDA_SAFE_CALL(hipEventDestroy(evt[i]));
	}

	CUDA_SAFE_CALL(hipHostFree(dB));
	CUDA_SAFE_CALL(hipHostFree(dA));
	CUDA_SAFE_CALL(hipHostFree(dC));
	CUDA_SAFE_CALL(hipHostFree(dAT));
	CUDA_SAFE_CALL(hipHostFree(dCT));

	/* Without prefetching with pinned memory begins  */
	
	CUDA_SAFE_CALL(hipHostMalloc(&dB,size, hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dA,(S*size/N), hipHostMallocDefault));
	CUDA_SAFE_CALL(hipHostMalloc(&dC,(S*size/N), hipHostMallocDefault));

	gettimeofday(&t1,0);
	CUDA_SAFE_CALL(hipMemcpy(dB,hB,size,hipMemcpyHostToDevice));
	for(LONG i=0; i< (N/S); i++){
		//cout << "Iteration " << i << endl;
	
		CUDA_SAFE_CALL(hipMemcpy(dA,hA+i*N*S,(S*size/N),hipMemcpyHostToDevice));
	
		//Execute the matrix multiplication kernel	
		gpuMM<<<grid,threadBlock>>>(dA,dB,dC,N);
	
		// Now copy the GPU result back to CPU
		CUDA_SAFE_CALL(hipMemcpy(C+i*N*S,dC,(S*size/N),hipMemcpyDeviceToHost));

	}
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	gettimeofday(&t2,0);
	timersub(&t2,&t1,&tnp);

	tt = (double) tnp.tv_sec + ((double) tnp.tv_usec/1.0e6);
	gflops = ( 1.0e-9 * 2.0 * N * N * N ) / tt; 
	//cout << "Without Prefetch : " << gflops << " Time : " << tt << " s"  << endl; 
	tnprepin = tt;

	CUDA_SAFE_CALL(hipHostFree(dB));
	CUDA_SAFE_CALL(hipHostFree(dA));
	CUDA_SAFE_CALL(hipHostFree(dC));

	/* Without prefetching without pinned memory begins  */
	
	CUDA_SAFE_CALL(hipMalloc(&dB,size));
	CUDA_SAFE_CALL(hipMalloc(&dA,(S*size/N)));
	CUDA_SAFE_CALL(hipMalloc(&dC,(S*size/N)));

	gettimeofday(&t1,0);
	CUDA_SAFE_CALL(hipMemcpy(dB,hB,size,hipMemcpyHostToDevice));
	for(LONG i=0; i< (N/S); i++){
		//cout << "Iteration " << i << endl;
	
		CUDA_SAFE_CALL(hipMemcpy(dA,hA+i*N*S,(S*size/N),hipMemcpyHostToDevice));
	
		//Execute the matrix multiplication kernel	
		gpuMM<<<grid,threadBlock>>>(dA,dB,dC,N);
	
		// Now copy the GPU result back to CPU
		CUDA_SAFE_CALL(hipMemcpy(C+i*N*S,dC,(S*size/N),hipMemcpyDeviceToHost));

	}
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	gettimeofday(&t2,0);
	timersub(&t2,&t1,&tnp);

	tt = (double) tnp.tv_sec + ((double) tnp.tv_usec/1.0e6);
	gflops = ( 1.0e-9 * 2.0 * N * N * N ) / tt; 
	//cout << "Without Prefetch : " << gflops << " Time : " << tt << " s"  << endl; 
	tnpre = tt;

	CUDA_SAFE_CALL(hipFree(dB));
	CUDA_SAFE_CALL(hipFree(dA));
	CUDA_SAFE_CALL(hipFree(dC));
#endif

	/* With Managed memory begins  */

	CUDA_SAFE_CALL(hipMallocManaged(&dA,size));
	CUDA_SAFE_CALL(hipMallocManaged(&dB,size));
	CUDA_SAFE_CALL(hipMallocManaged(&dC,size));

	dim3 threadBlock_um(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid_um(K,K);

	// Initialize matrices
	for (LONG j=0; j<N; j++){
	    for (LONG i=0; i<N; i++){
	    	dA[j*N+i] = hA[j*N+i];
		dB[j*N+i] = hB[j*N+i];
	    }
	}
	
	gettimeofday(&t1,0);

	for(LONG i=0; i< (N/S); i++){
		gpuMM<<<grid,threadBlock>>>(dA+i*N*S,dB,dC+i*N*S,N);
	}

	//gpuMM_um<<<grid_um,threadBlock_um>>>(dA,dB,dC,N);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	
	gettimeofday(&t2,0);
	timersub(&t2,&t1,&tp);

	tt = (double) tp.tv_sec + ((double) tp.tv_usec/1.0e6);
	gflops = ( 1.0e-9 * 2.0 * N * N * N ) / tt; 
	//cout << "Managed : " << gflops << endl; 
	tmgm = tt;

	CUDA_SAFE_CALL(hipFree(dA));
	CUDA_SAFE_CALL(hipFree(dB));
	CUDA_SAFE_CALL(hipFree(dC));

#if 0
	// Now do the matrix multiplication on the CPU
	double sum;
	for (LONG row=0; row<N; row++){
		for (LONG col=0; col<N; col++){
			sum = 0.f;
			for (LONG n=0; n<N; n++){
				sum += hA[row*N+n]*hB[n*N+col];
			}
			hC[row*N+col] = sum;
		}
	}
#endif

	cout << N << "x" << N << " " << tpre << " " << tprenk << " " << tnprepin << " " << tnpre << " " << tmgm << endl;

	delete [] hA;
	delete [] hB;
	delete [] hC;
	delete [] C;

	}

	cout << "Finished." << endl;

	return 0;
}
