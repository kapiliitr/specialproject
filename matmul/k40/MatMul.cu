
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>

using namespace std;

#define CUDA_SAFE_CALL( err ) (safe_call(err, __LINE__))
#define BLOCK_SIZE 32

typedef unsigned long long int LONG;

void safe_call(hipError_t ret, int line)
{
	if(ret!=hipSuccess)
	{
		cout << "Error at line " << line << " : " << hipGetErrorString(ret) << endl;
		exit(-1);
	}
}

__global__ void gpuMM(double *A, double *B, double *C, LONG N)
{
	// Matrix multiplication for NxN matrices C=A*B
	// Each thread computes a single element of C
	LONG row = blockIdx.y*blockDim.y + threadIdx.y;
	LONG col = blockIdx.x*blockDim.x + threadIdx.x;

	double sum = 0.f;
	for (LONG n = 0; n < N; ++n)
	    sum += A[row*N+n]*B[n*N+col];

	C[row*N+col] = sum;
}

int main(int argc, char *argv[])
{
	struct timeval t1,t2, tp;
	double tt, gflops;

	// Perform matrix multiplication C = A*B
	// where A, B and C are NxN matrices
	// Restricted to matrices where N = K*BLOCK_SIZE;
	LONG N,K;
	cin >> K;
	N = K*BLOCK_SIZE;

	CUDA_SAFE_CALL(hipSetDevice(0));

	cout << "Executing Matrix Multiplcation" << endl;
	cout << "Matrix size: " << N << "x" << N << endl;

#if 0
	// Allocate memory on the host
	double *hA,*hB,*hC;
	hA = new double[N*N];
	hB = new double[N*N];
	hC = new double[N*N];

	// Initialize matrices on the host
	for (LONG j=0; j<N; j++){
	    for (LONG i=0; i<N; i++){
	    	hA[j*N+i] = 2.f*(j+i);
			hB[j*N+i] = 1.f*(j-i);
	    }
	}
#endif

	// Allocate memory on the device
	LONG size = N*N*sizeof(double);	// Size of the memory in bytes
	double *dA,*dB,*dC;
	CUDA_SAFE_CALL(hipMallocManaged(&dA,size));
	CUDA_SAFE_CALL(hipMallocManaged(&dB,size));
	CUDA_SAFE_CALL(hipMallocManaged(&dC,size));

	cout << "Memory allocated on device memory." << endl;

	// Initialize matrices
	for (LONG j=0; j<N; j++){
	    for (LONG i=0; i<N; i++){
	    	dA[j*N+i] = 2.f*(j+i);
		dB[j*N+i] = 1.f*(j-i);
	    }
	}

	dim3 threadBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(K,K);
	
	gettimeofday(&t1,0);

	// Copy matrices from the host to device
	//CUDA_SAFE_CALL(cudaMemcpy(dA,hA,size,cudaMemcpyHostToDevice));
	//CUDA_SAFE_CALL(cudaMemcpy(dB,hB,size,cudaMemcpyHostToDevice));
	
	//Execute the matrix multiplication kernel
	
	gpuMM<<<grid,threadBlock>>>(dA,dB,dC,N);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	
	gettimeofday(&t2,0);
	timersub(&t2,&t1,&tp);

	tt = (double) tp.tv_sec + ((double) tp.tv_usec/1.0e6);
	gflops = ( 1.0e-9 * 2.0 * N * N * N ) / tt; 
	cout << "Managed : " << gflops << endl; 

#if 0
	// Now do the matrix multiplication on the CPU
	double sum;
	for (LONG row=0; row<N; row++){
		for (LONG col=0; col<N; col++){
			sum = 0.f;
			for (LONG n=0; n<N; n++){
				sum += hA[row*N+n]*hB[n*N+col];
			}
			hC[row*N+col] = sum;
		}
	}

	// Allocate memory to store the GPU answer on the host
	double *C;
	C = new double[N*N];
	
	// Now copy the GPU result back to CPU
	CUDA_SAFE_CALL(cudaMemcpy(C,dC,size,cudaMemcpyDeviceToHost));
	
	// Check the result and make sure it is correct
	for (LONG row=0; row<N; row++){
		for (LONG col=0; col<N; col++){
			if ( C[row*N+col] != hC[row*N+col] ){
				cout << "Wrong answer!" << endl;
				row = col = N;
			}
		}
	}

#endif
		
	cout << "Finished." << endl;

	CUDA_SAFE_CALL(hipFree(dA));
	CUDA_SAFE_CALL(hipFree(dB));
	CUDA_SAFE_CALL(hipFree(dC));

	return 0;
}
